#include "hip/hip_runtime.h"

%%cu


#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

// Activation function (sigmoid)
__device__ double sigmoid(double x) {
    return 1.0 / (1.0 + exp(-x));
}

// Derivative of the activation function (sigmoid derivative)
__device__ double sigmoidDerivative(double x) {
    double s = sigmoid(x);
    return s * (1.0 - s);
}

// CUDA kernel for weight update
__global__ void backpropagationKernel(double* input, double* hiddenLayer, double* outputLayer,
                                      double* target, double* weightsInputHidden, double* weightsHiddenOutput,
                                      double learningRate, int inputSize, int hiddenSize, int outputSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < inputSize && col < hiddenSize) {
        double gradient = 0.0;
        for (int i = 0; i < outputSize; ++i) {
            double outputError = target[i] - outputLayer[i];
            gradient += outputError * sigmoidDerivative(outputLayer[i]) * weightsHiddenOutput[col * outputSize + i];
        }
        weightsInputHidden[row * hiddenSize + col] += learningRate * gradient * sigmoidDerivative(hiddenLayer[col]) * input[row];
    }

    if (row < hiddenSize && col < outputSize) {
        double outputError = target[col] - outputLayer[col];
        weightsHiddenOutput[row * outputSize + col] += learningRate * outputError * sigmoidDerivative(outputLayer[col]) * hiddenLayer[row];
    }
}

class NeuralNetwork {
private:
    // ... (rest of the code remains the same)

public:
    // ... (rest of the code remains the same)

    // CUDA backpropagation pass
    void trainCUDA(const std::vector<double>& input, const std::vector<double>& target) {
        // Check if the input and target size match the input and output layer size, respectively
        if (input.size() != inputSize || target.size() != outputSize) {
            std::cerr << "Input or target size does not match the input or output layer size." << std::endl;
            return;
        }

        // Allocate device memory
        double* d_input;
        double* d_hiddenLayer;
        double* d_outputLayer;
        double* d_target;
        double* d_weightsInputHidden;
        double* d_weightsHiddenOutput;

        hipMalloc((void**)&d_input, inputSize * sizeof(double));
        hipMalloc((void**)&d_hiddenLayer, hiddenSize * sizeof(double));
        hipMalloc((void**)&d_outputLayer, outputSize * sizeof(double));
        hipMalloc((void**)&d_target, outputSize * sizeof(double));
        hipMalloc((void**)&d_weightsInputHidden, inputSize * hiddenSize * sizeof(double));
        hipMalloc((void**)&d_weightsHiddenOutput, hiddenSize * outputSize * sizeof(double));

        // Copy data to device
        hipMemcpy(d_input, input.data(), inputSize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_hiddenLayer, hiddenLayer.data(), hiddenSize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_outputLayer, outputLayer.data(), outputSize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_target, target.data(), outputSize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_weightsInputHidden, weightsInputHidden.data(), inputSize * hiddenSize * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_weightsHiddenOutput, weightsHiddenOutput.data(), hiddenSize * outputSize * sizeof(double), hipMemcpyHostToDevice);

        // Define the grid and block dimensions
        dim3 threadsPerBlock(32, 1);
        dim3 numBlocks((hiddenSize + threadsPerBlock.x - 1) / threadsPerBlock.x, 1);

        // Call the CUDA kernel
        backpropagationKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_hiddenLayer, d_outputLayer, d_target,
                                                              d_weightsInputHidden, d_weightsHiddenOutput,
                                                              learningRate, inputSize, hiddenSize, outputSize);

        // Copy updated weights back to the host
        hipMemcpy(weightsInputHidden.data(), d_weightsInputHidden, inputSize * hiddenSize * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(weightsHiddenOutput.data(), d_weightsHiddenOutput, hiddenSize * outputSize * sizeof(double), hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_input);
        hipFree(d_hiddenLayer);
        hipFree(d_outputLayer);
        hipFree(d_target);
        hipFree(d_weightsInputHidden);
        hipFree(d_weightsHiddenOutput);
    }
};

int main() {
    // ... (rest of the code remains the same)
}
